#include "hip/hip_runtime.h"
/*
 * Copyright 2014 Google Inc. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *    http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 * ---------------------------------------------------------------------------
 * Copyright 2014 Nervana Systems Inc.  All rights reserved.
 *
 * * Added timing diagnostics
 * ---------------------------------------------------------------------------
 */

#include <vector>
#include <iostream> 
#include <string>
#include <set>
#include <map>

#include "../../nvmatrix/include/nvmatrix.cuh"
#include "../../nvmatrix/include/nvmatrix_operators.cuh"
#include "../../util/include/matrix.h"
#include "../include/convnet.cuh"
#include "../include/util.cuh"

using namespace std;

/* 
 * =======================
 * ConvNet
 * =======================
 */
ConvNet::ConvNet(PyObject* layerParams, intv& deviceIDs,
                 int minibatchSize, bool conserveMem) : Thread(true) {
    _deviceIDs = deviceIDs;
    _data = NULL;
    _bufferData = NULL;
    _bufferMinibatchIdx = -1;
    _bufferPassIdx = -1;
    _trainingProgress = 0;
    _totalPassesDone = 0;
    _conserveMem = conserveMem;
    _sync = new ThreadSynchronizer(deviceIDs.size() + 1);
    PyObjectV* layerList = pyDictGetValues(layerParams);
    std::sort(layerList->begin(), layerList->end(), LayerIDComparator());

    
    _dataCopyPD = new PipeDispenserBlocking(DIVUP(_deviceIDs.size(),2)); // hard-coded for now

    initDataLayers(layerList);
    initGPUThreads(layerList);
    connectReplicas();              // Connect replicas to one another
    connectChildren(layerParams);   // Connect forward/backward links in graph
    _numFwdTerminal = 0;
    // Execute post-initialization stuff
    for (NameReplicaLayerMap::iterator it = _layerMap.begin(); it != _layerMap.end(); ++it) {
        for (int r = 0; r < it->second.size(); r++) {
            _numFwdTerminal += it->second[r]->getNext().size() == 0;
            if (it->second[r]->getNext().size() == 0) {
                printf("Fwd terminal: %s\n", it->second[r]->getName().c_str());
            }
            it->second[r]->postInit();
        }
    }

    // Find and count the terminal nodes in the backward pass
    for (int p = 0; p < getNumPasses(); p++) {
        set<Layer*> visited;
        _numBwdTerminal[p] = 0;
        for (int t = 0; t < _convNetThreads.size(); t++) {
            vector<CostLayer*>& cl = _convNetThreads[t]->getCostLayers();
            for (int c = 0; c < cl.size(); c++) {
                findBwdTerminal(*cl[c], visited, _numBwdTerminal[p], p);
            }
        }
    }

    _dp = new DataProvider(minibatchSize);
//    Py_DECREF(layerList);
    delete layerList;
}

ConvNet::~ConvNet() {
    for (vector<ConvNetThread*>::const_iterator it = _convNetThreads.begin(); it != _convNetThreads.end(); ++it) {
        (*it)->getMessageQueue().enqueue(new Message(EXIT_CONVNET));
        (*it)->join();
        delete *it;
    }
    for (DataLayerVector::const_iterator it = _dataLayers.begin(); it != _dataLayers.end(); ++it) {
        delete *it;
    }
    for (intv::const_iterator it = _deviceIDs.begin(); it != _deviceIDs.end(); ++it) {
        DEVICE_MEMORY_MANAGER::destroyInstance(*it);
    }
    HOST_MEMORY_MANAGER::destroyInstance();
    delete _sync;
    delete _dataCopyPD;
    delete _dp;
}

void ConvNet::stop() {
    getWorkerQueue().enqueue(new ExitWorker(*this));
    join();
}

PipeDispenser& ConvNet::getDataCopyPD() {
    return *_dataCopyPD;
}

void ConvNet::initDataLayers(PyObjectV* layerList) {
    for (int i = 0; i < layerList->size(); i++) {
        PyObject* paramsDict = layerList->at(i);
        std::string layerType = pyDictGetString(paramsDict, "type");

        if (layerType == "data") {
            int numReplicas = pyDictGetInt(paramsDict, "numReplicas");
            for (int r = 0; r < numReplicas; ++r) {
                DataLayer* dataLayer = new DataLayer(this, paramsDict, r);
                _dataLayers.push_back(dataLayer);
                _layerMap[dataLayer->getName()][r] = dataLayer;
            }
        }
    }
}

void ConvNet::initGPUThreads(PyObjectV* layerList) {
    // Initialize GPU worker threads
    for (int i = 0; i < _deviceIDs.size(); ++i) {
        ConvNetThread* cng = new ConvNetThread(layerList, _deviceIDs[i], i, this);
        _convNetThreads.push_back(cng);
        for (NameLayerMap::iterator it = cng->getLayerMap().begin(); it != cng->getLayerMap().end(); ++it) {
            const std::string& name = it->first;
            Layer* layer = it->second;
            _layerMap[name][layer->getReplicaID()] = layer;
        }
    }
}

void ConvNet::connectReplicas() {
    _numReplicasMax = 0;
    _numReplicasMin = 1 << 16;
    for (NameReplicaLayerMap::iterator it = _layerMap.begin(); it != _layerMap.end(); ++it) {
        _numReplicasMax = max(_numReplicasMax, int(it->second.size()));
        _numReplicasMin = min(_numReplicasMin, int(it->second.size()));
        for (map<int,Layer*>::iterator it2 = it->second.begin(); it2 != it->second.end(); ++it2) {
            Layer& l1 = *it2->second;
            for (map<int,Layer*>::iterator it3 = it->second.begin(); it3 != it->second.end(); ++it3) {
                Layer& l2 = *it3->second;
                l1.addReplica(l2);
            }
        }
    }
}

void ConvNet::connectChildren(PyObject* layerParams) {
    for (NameReplicaLayerMap::iterator it = _layerMap.begin(); it != _layerMap.end(); ++it) {
        PyObject* paramsDict = PyDict_GetItemString(layerParams, it->first.c_str());
        PyObject* inputList = PyDict_GetItemString(paramsDict, "inputs");
        if (inputList != NULL) {
            // Iterate over "replicas" of this layer
            int numReplicas = _layerMap[it->first].size();
            for (int i = 0; i < PyList_GET_SIZE(inputList); i++) {
                std::string inputName = PyString_AsString(PyList_GetItem(inputList, i));
                int numReplicasPrev = _layerMap[inputName].size();
                // How many replicas from the previous layer must this layer be connected to?
                int numInputReplicas = numReplicasPrev / numReplicas;
                for (int r = 0; r < numReplicas; r++) {
                    for (int rp = r, ridx = 0; ridx < numInputReplicas; rp += numReplicas, ridx++) {
                        it->second[r]->addPrev(*_layerMap[inputName][rp], ridx);
                        _layerMap[inputName][rp]->addNext(*it->second[r]);
                    }
                }
            }
        }
    }
}

void ConvNet::findBwdTerminal(Layer& l, set<Layer*>& visited, int& terminal, int passIdx) {
    if (visited.count(&l) == 0) {
        visited.insert(&l);
        if (l.isGradConsumer()) {
            bool hasPrevConsumer = false;
            if (l.getPrev().size() > 0) {
                for (int i = 0; i < l.getPrev()[0].size(); i++) {
                    // Looking only at 0th replica is fine to see if you have
                    // grad consumers below you.
                    hasPrevConsumer |= l.getPrev()[0][i]->isGradConsumer();
                }
            }
            if (!hasPrevConsumer || !l.isGradProducer() || (passIdx + 1 < l.getNumReplicasPrev() && l.getNumReplicasPrev() > l.getNumReplicas())) {
                terminal++;
                l.setBwdTerminal(passIdx);
                printf("found bwd terminal %s[%d] in passIdx=%d\n", l.getName().c_str(), l.getReplicaID(), passIdx);
            } else if (l.isGradProducer()) {
                for (int r = 0; r < l.getPrev().size(); r++) {
                    for (int i = 0; i < l.getPrev()[r].size(); i++) {
                        findBwdTerminal(*l.getPrev()[r][i], visited, terminal, passIdx);
                    }
                }
            }
        }
    }
}

void* ConvNet::run() {
    for (vector<ConvNetThread*>::const_iterator it = _convNetThreads.begin(); it != _convNetThreads.end(); ++it) {
        (*it)->start();
    }
    // The manager thread defaults to using the GPU of the first worker.
    // Put more logic here if this is inappropriate.
    NVMatrix::setDeviceID(_convNetThreads[0]->getDeviceID());
    copyToGPU();
    bool exit = false;
    while (!exit) {
        Worker* worker = _workerQueue.dequeue();
        exit = worker->run();
        delete worker;
    }

    return NULL;
}

Queue<Worker*>& ConvNet::getWorkerQueue() {
    return _workerQueue;
}

Queue<WorkResult*>& ConvNet::getResultQueue() {
    return _resultQueue;
}

DataProvider& ConvNet::getDataProvider() {
    return *_dp;
}

Layer& ConvNet::getLayer(std::string& name, int replicaID) {
    return *_layerMap[name][replicaID];
}

void ConvNet::sendMessage(MESSAGES msg, bool sync) {
    sendMessage(new Message(msg), sync);
}

void ConvNet::sendMessage(Message* msg, bool sync) {
    for (int i = 0; i < _convNetThreads.size(); i++) {
        _convNetThreads[i]->getMessageQueue().enqueue(msg->clone());
    }

    delete msg;

    if (sync) {
        syncWithChildren();
    }
}

void ConvNet::copyToCPU() {
    sendMessage(COPY_TO_CPU, true);
}

void ConvNet::copyToGPU() {
    sendMessage(COPY_TO_GPU, false);
}

void ConvNet::updateWeights(int passIdx) {
    sendMessage(UPDATE_WEIGHTS, true);
    sendMessage(CONSTRAIN_WEIGHTS, true);
}

void ConvNet::reset(int passIdx) {
    sendMessage((passIdx % getNumPasses()) == 0 ? RESET : RESET_PASS_IDX, false);
}

void ConvNet::reset() {
    reset(0);
}

// Fprop given data
void ConvNet::fprop(CPUData& data, int passIdx, PASS_TYPE passType) {
    reset(passIdx);
    // This is necessary because setData below could delete data. If there's
    // an outstanding copy request, this'll cause a segfault.
    for (int i = 0; i < _dataLayers.size(); i++) {
        _dataLayers[i]->waitForCopyFinish();
    }

    setData(data, passIdx);
    for (int i = 0; i < _dataLayers.size(); i++) {
        _dataLayers[i]->fprop(passType, passIdx, false);
    }
    waitForTerminals(_numFwdTerminal, FPROP_TERMINAL);
}

// Fprop given minibatch idx
void ConvNet::fprop(int miniIdx, int passIdx, PASS_TYPE passType) {
    reset(passIdx);

    bool fromBuffer = miniIdx == _bufferMinibatchIdx && passIdx == _bufferPassIdx;
    if (!fromBuffer) {
        // This is necessary because setData below could delete data. If there's
        // an outstanding copy request, this'll cause a segfault.
        for (int i = 0; i < _dataLayers.size(); i++) {
            _dataLayers[i]->waitForCopyFinish();
        }

        setData(_dp->getMinibatch(miniIdx), passIdx);

    } else {
        setDataFromBuffer();
    }
    for (int i = 0; i < _dataLayers.size(); i++) {
        _dataLayers[i]->fprop(passType, passIdx, fromBuffer);
    }

    if (passIdx == getNumPasses() - 1) {
        // Do double-buffering from next minibatch from the DataProvider
        setBuffer(miniIdx == _dp->getNumMinibatches() - 1 ? NULL : &_dp->getMinibatch(miniIdx + 1), miniIdx + 1, 0);
    } else {
        // Do double-buffering from next microbatch within current minibatch
        setBuffer(_data, miniIdx, passIdx + 1);
    }

    waitForTerminals(_numFwdTerminal, FPROP_TERMINAL);
}

void ConvNet::setDataFromBuffer() {
    if (_bufferData != _data) {
        delete _data;
    }
    _data = _bufferData;
    _bufferData = NULL;
    _bufferMinibatchIdx = -1;
    _bufferPassIdx = -1;
}

void ConvNet::setData(CPUData& data, int passIdx) {
    bool same = _data == _bufferData;
    if (&data != _data) {
        delete _data;
    }
    if (&data != _bufferData && !same) {
        delete _bufferData;
        _bufferData = NULL;
        _bufferMinibatchIdx = -1;
        _bufferPassIdx = -1;
    }
    _data = &data;
    for (int i = 0; i < _dataLayers.size(); i++) {
        _dataLayers[i]->copyData(*_data, false, passIdx);
    }
}

void ConvNet::setBuffer(CPUData* bufferData, int bufferMinibatchIdx, int bufferPassIdx) {
    _bufferData = bufferData;
    _bufferMinibatchIdx = bufferMinibatchIdx;
    _bufferPassIdx = bufferPassIdx;
    if (bufferData != NULL) {
        for (int i = 0; i < _dataLayers.size(); i++) {
            _dataLayers[i]->copyData(*_bufferData, true, bufferPassIdx);
        }
    }
}

CPUData& ConvNet::getData() {
    assert(_data != NULL);
    return *_data;
}

void ConvNet::bprop(int passIdx, PASS_TYPE passType) {
    _totalPassesDone++;
    sendMessage(new BpropStartMessage(passType, passIdx), false);
    waitForTerminals(_numBwdTerminal[passIdx], BPROP_TERMINAL);
    reset(passIdx + 1);
}

void ConvNet::waitForTerminals(int numMsgs, MESSAGES msgType) {
    for (int rcvd = 0; rcvd < numMsgs; rcvd++) {
        Message* m = _msgQueue.dequeue();
        assert(m->getType() == msgType);
        delete m;
    }
}

// Same as getCost() but adds results to given cost and returns it
Cost& ConvNet::getCost(Cost& cost) {
    Cost &tmp = getCost();
    cost += tmp;
    delete &tmp;
    return cost;
}

Cost& ConvNet::getCost() {
    Cost& cost = *new Cost();
    for (int t = 0; t < _convNetThreads.size(); t++) {
        Cost& tcost = _convNetThreads[t]->getCost();
        cost += tcost;
        delete &tcost;
    }
    return cost;
}

double ConvNet::getCostValue() {
    Cost& cost = getCost();
    double val = cost.getValue();
    delete &cost;
    return val;
}

Queue<Message*>& ConvNet::getMessageQueue() {
    return _msgQueue;
}

intv& ConvNet::getDeviceIDs() {
    return _deviceIDs;
}

ThreadSynchronizer& ConvNet::getSync() {
    return *_sync;
}

void ConvNet::syncWithChildren() {
    sendMessage(SYNC, false);
    _sync->sync();
}

int ConvNet::getTotalPassesDone() {
    return _totalPassesDone;
}

int ConvNet::getMinibatchSize() {
    return _dp->getMinibatchSize();
}

int ConvNet::getNumReplicasMax() {
    return _numReplicasMax;
}

int ConvNet::getNumReplicasMin() {
    return _numReplicasMin;
}

int ConvNet::getNumPasses() {
    return _numReplicasMax / _numReplicasMin;
}

void ConvNet::setTrainingProgress(double progress) {
    _trainingProgress = progress;
}

double ConvNet::getTrainingProgress() const {
    return _trainingProgress;
}

bool ConvNet::isConserveMemory() {
    return _conserveMem;
}

/*
 * Gradient checking stuff
 */
void ConvNet::checkGradients() {
    _numFailures = 0;
    _numTests = 0;
    _baseErr = 0;
    for (int p = 0; p < getNumPasses(); ++p) {
        fprop(0, p, PASS_GC);
        _baseErr += getCostValue();
        bprop(p, PASS_GC);
    }
    // We call grad check only on the first replica,
    // but because weights are aware of their fellow replicas,
    // we can simultaneously perturb the weights of all
    // replicas.
    for (NameReplicaLayerMap::iterator it = _layerMap.begin(); it != _layerMap.end(); ++it) {
        map<int, Layer*>& layers = it->second;
        if (layers[0]->getDeviceID() >= 0 /*&& (layers[0]->getName() == "fc10")*/) { // If layer on GPU (data layers aren't)
            layers[0]->checkGradient();
        }
    }

    cout << "------------------------" << endl;
    if (_numFailures > 0) {
        cout << _numFailures << "/" << _numTests << " TESTS FAILED" << endl;
    } else {
        cout << "ALL " << _numTests << " TESTS PASSED" << endl;
    }
}

// Copies to all replicas
void ConvNet::checkGradient_copyWeightsToGPU(Matrix& weightsCPU, Weights& weights) {
    int d = NVMatrix::getDeviceID();
    for (map<int, Weights*>::const_iterator it = weights.getReplicas().begin(); it != weights.getReplicas().end(); ++it) {
        NVMatrix::setDeviceID(it->second->getDeviceID());
        it->second->getW().copyFromHost(weightsCPU);
    }
    NVMatrix::setDeviceID(d);
}

/*
 * name: weight matrix name
 * eps: finite difference step
 */
bool ConvNet::checkGradient(const std::string& name, float eps, Weights& weights) {
    Matrix numGrad(weights.getNumRows(), weights.getNumCols());
    Matrix diff(numGrad);
    numGrad.apply(Matrix::ZERO);
    Matrix weightsCPU;

    weights.getW().copyToHost(weightsCPU, true);

    for(int i = 0; i < weights.getNumRows(); i++) {
        for (int j = 0; j < weights.getNumCols(); j++) {
            float v = weightsCPU(i,j);
            weightsCPU(i,j) += eps;

            checkGradient_copyWeightsToGPU(weightsCPU, weights);

            weightsCPU(i,j) = v;
            double err = 0;
            for (int p = 0; p < getNumPasses(); ++p) {
//                printf("trying fprop %d\n", p);
                fprop(0, p, PASS_GC);
//                printf("    success\n");
                err += getCostValue();
            }
            numGrad(i,j) = (err - _baseErr) / (_data->getNumCases() * eps);
            if (isnan((double)numGrad(i,j)) || isinf((double)numGrad(i,j))) {
                cout << "Numerical computation produced nan or inf when checking '" << name << "': " << numGrad(i,j) << endl;
                cout << "Consider reducing the sizes of the weights or finite difference steps." << endl;
                cout << "Exiting." << endl;
                exit(1);
            }
            checkGradient_copyWeightsToGPU(weightsCPU, weights);
        }
    }
    Matrix gradCPU;
    NVMatrix::setDeviceID(weights.getDeviceID());
    map<int,NVMatrix*> mats;
    for (map<int, Weights*>::const_iterator it = weights.getReplicas().begin(); it != weights.getReplicas().end(); ++it) {
        mats[it->first] = &it->second->getGrad();
    }
    weights.getReducer().reduce(mats, 1, false);

    weights.getGrad().copyToHost(gradCPU, true);
    gradCPU.scale(-1.0 / _data->getNumCases());
    float analNorm = gradCPU.norm();
    float numNorm = numGrad.norm();
    numGrad.subtract(gradCPU, diff);
    float relErr = diff.norm() / analNorm;
    bool fail = relErr >= GC_REL_ERR_THRESH;
    if (fail || !GC_SUPPRESS_PASSES) {
        cout << "========================" << endl;
        printf("(%s) %s GRADIENT CHECK\n", fail ? "****FAIL****" : "PASS", name.c_str());
        cout << "========================" << endl;
        cout << "Analytic:" << endl;
        gradCPU.print(0, 6, 0, 4);
        cout << "Numeric:" << endl;
        numGrad.print(0, 6, 0, 4);
        printf("Analytic norm: %e\n", analNorm);
        printf("Numeric norm:  %e\n", numNorm);
        printf("Relative error: %e\n", relErr);
    }
    _numTests++;
    _numFailures += fail;
    return fail;
}

/* 
 * =======================================================================================================
 * ConvNetThread
 * =======================================================================================================
 */
ConvNetThread::ConvNetThread(PyObjectV* layerList, int deviceID, int deviceIdx, ConvNet* convNet)
    : Thread(true, getDeviceCPUs(deviceID)), _deviceID(deviceID), _convNet(convNet) {
    try {
        int numLayers = layerList->size();

        for (int i = 0; i < numLayers; i++) {
            PyObject* paramsDict = layerList->at(i);
            std::string layerType = pyDictGetString(paramsDict, "type");
            if (layerType != "data") {
                intv& gpus = *pyDictGetIntV(paramsDict, "gpu");
                int rid = indexOf(gpus, deviceIdx);
                if (rid >= 0) {
                    initLayer(paramsDict, rid);
                }
                delete &gpus;
            }
        }
    } catch (std::string& s) {
        cout << "Error creating ConvNet: " << s << endl;
        exit(1);
    }
}

ConvNetThread::~ConvNetThread() {
    NVMatrix::setDeviceID(_deviceID);
    NVMatrix::destroyCublas();
    NVMatrix::destroyRandom();
    for (NameLayerMap::const_iterator it = _nameLayerMap.begin(); it != _nameLayerMap.end(); ++it) {
        delete it->second;
    }
    _nameLayerMap.clear();
}

void ConvNetThread::startTimer() {
    NVMatrix::syncStream();
    _timer.start();
}

double ConvNetThread::stopTimer() {
    NVMatrix::syncStream();
    return _timer.stop();
}

void ConvNetThread::initLayer(PyObject* paramsDict, int replicaID) {
    std::string type = pyDictGetString(paramsDict, "type");
    std::string name = pyDictGetString(paramsDict, "name");
    if (type == "fc") {
        _nameLayerMap[name] = new FCLayer(this, paramsDict, replicaID, false);
    } else if (type == "sfc") {
        _nameLayerMap[name] = new SplitFCLayer(this, paramsDict, replicaID, false);
    } else if (type == "conv") {
        _nameLayerMap[name] = new ConvLayer(this, paramsDict, replicaID);
    } else if (type == "local") {
        _nameLayerMap[name] = new LocalUnsharedLayer(this, paramsDict, replicaID);
    } else if (type == "pool") {
        _nameLayerMap[name] = &PoolLayer::make(this, paramsDict, replicaID);
    } else if (type == "cmpool") {
        _nameLayerMap[name] = &CrossMapPoolLayer::make(this, paramsDict, replicaID);
    } else if (type == "rnorm") {
        _nameLayerMap[name] = new ResponseNormLayer(this, paramsDict, replicaID);
    } else if (type == "cmrnorm") {
        _nameLayerMap[name] = new CrossMapResponseNormLayer(this, paramsDict, replicaID);
    } else if (type == "cnorm") {
        _nameLayerMap[name] = new ContrastNormLayer(this, paramsDict, replicaID);
    } else if (type == "softmax") {
        _nameLayerMap[name] = new SoftmaxLayer(this, paramsDict, replicaID);
    } else if (type == "eltsum") {
        _nameLayerMap[name] = new EltwiseSumLayer(this, paramsDict, replicaID);
    } else if (type == "eltmax") {
        _nameLayerMap[name] = new EltwiseMaxLayer(this, paramsDict, replicaID);
    } else if (type == "neuron") {
        _nameLayerMap[name] = new NeuronLayer(this, paramsDict, replicaID);
    } else if (type == "nailbed") {
        _nameLayerMap[name] = new NailbedLayer(this, paramsDict, replicaID);
    } else if (type == "blur") {
        _nameLayerMap[name] = new GaussianBlurLayer(this, paramsDict, replicaID);
    } else if (type == "href") {
        _nameLayerMap[name] = new HorizontalReflectionLayer(this, paramsDict, replicaID);
    } else if (type == "resize") {
        _nameLayerMap[name] = new ResizeLayer(this, paramsDict, replicaID);
    } else if (type == "rgb2yuv") {
        _nameLayerMap[name] = new RGBToYUVLayer(this, paramsDict, replicaID);
    } else if (type == "rgb2lab") {
        _nameLayerMap[name] = new RGBToLABLayer(this, paramsDict, replicaID);
    } else if (type == "rscale") {
        _nameLayerMap[name] = new RandomScaleLayer(this, paramsDict, replicaID);
    } else if (type == "crop") {
        _nameLayerMap[name] = new CropLayer(this, paramsDict, replicaID);
    } else if (type == "concat") {
        _nameLayerMap[name] = new ConcatenationLayer(this, paramsDict, replicaID);
    } else if (type == "pass") {
        _nameLayerMap[name] = new PassThroughLayer(this, paramsDict, replicaID);
    } else if (type == "dropout") {
        _nameLayerMap[name] = new DropoutLayer(this, paramsDict, replicaID);
    } else if (type == "dropout2") {
        _nameLayerMap[name] = new Dropout2Layer(this, paramsDict, replicaID);
    } else if (strncmp(type.c_str(), "cost.", 5) == 0) {
        CostLayer *c = &CostLayer::make(this, paramsDict, type, replicaID);
        _nameLayerMap[name] = c;
        _costs.push_back(c);
    } else {
        throw std::string("Unknown layer type ") + type;
    }
}

/*
 * This executes in a new CPU thread so it's OK to initialize CUDA stuff here. 
 */
void ConvNetThread::initCuda() { 
    NVMatrix::setDeviceID(_deviceID);
    checkCudaErrors(hipDeviceSetCacheConfig(hipFuncCachePreferShared));
    for (int i = 0; i < _convNet->getDeviceIDs().size(); i++) {
        int d = _convNet->getDeviceIDs()[i];
        if (d != _deviceID) {
            if (NVMatrix::canAccessPeer(_deviceID, d)) {
                printf("Enabling peer access GPU %d --> GPU %d\n", NVMatrix::getDeviceID(), d);
                checkCudaErrors(hipDeviceEnablePeerAccess(d, 0));
            } else {
                printf("No peer access GPU %d -->  GPU %d\n", _deviceID, d);
            }
        }
    }
//    NVMatrix::syncStream();
    NVMatrix::initCublas();
    NVMatrix::initRandom(/*7*/);
    srand(time(0));
}

// (u) helper function for timing stuff
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
//

void* ConvNetThread::run() {
    initCuda();
    bool exit = false;
    // (u) setting up some timing stuff
    bool timingdump = false;
    float time;
    float time0=0;
    float time1=0;
    float time2=0;
    float time3=0;
    float time4=0;
    float time5=0;
    float time6=0;
    float time7=0;
    float time8=0;
    float time9=0;
    hipEvent_t start, stop;
    //
    while (!exit) {
        Message* m = _msgQueue.dequeue();
        if (timingdump) { printf("."); }
        if (m->getType() == FPROP_READY) {
            if (timingdump) { printf("0");
            HANDLE_ERROR( hipEventCreate(&start) ); HANDLE_ERROR( hipEventCreate(&stop) ); HANDLE_ERROR( hipEventRecord(start, 0) );}
            FpropMessage* msg = static_cast<FpropMessage*>(m);
            msg->getToLayer().fprop(msg->getPassType(), msg->getPassIdx());
            if (timingdump) { HANDLE_ERROR( hipEventRecord(stop, 0) ); HANDLE_ERROR( hipEventSynchronize(stop) ); HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) ); time0+=time;}
        } else if (m->getType() == BPROP_READY) {
            if (timingdump) { printf("1");
            HANDLE_ERROR( hipEventCreate(&start) ); HANDLE_ERROR( hipEventCreate(&stop) ); HANDLE_ERROR( hipEventRecord(start, 0) );}
            BpropMessage* msg = static_cast<BpropMessage*>(m);
            msg->getToLayer().incRcvdBInputMsgs();
            msg->getToLayer().bprop(msg->getPassType(), msg->getPassIdx());
            if (timingdump) { HANDLE_ERROR( hipEventRecord(stop, 0) ); HANDLE_ERROR( hipEventSynchronize(stop) ); HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) ); time1+=time;} 
        } else if (m->getType() == BPROP_START) {
            if (timingdump) { printf("2");
            HANDLE_ERROR( hipEventCreate(&start) ); HANDLE_ERROR( hipEventCreate(&stop) ); HANDLE_ERROR( hipEventRecord(start, 0) );}
            BpropStartMessage* msg = static_cast<BpropStartMessage*>(m);
            for (int i = 0; i < _costs.size(); i++) {
                dynamic_cast<Layer*>(_costs[i])->bprop(msg->getPassType(), msg->getPassIdx());
            }
            if (timingdump) { HANDLE_ERROR( hipEventRecord(stop, 0) ); HANDLE_ERROR( hipEventSynchronize(stop) ); HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) ); time2+=time;}
        } else if (m->getType() == SYNC) {
            if (timingdump) { printf("3");
            HANDLE_ERROR( hipEventCreate(&start) ); HANDLE_ERROR( hipEventCreate(&stop) ); HANDLE_ERROR( hipEventRecord(start, 0) );}
            NVMatrix::syncStream();
            _convNet->getSync().sync();
            if (timingdump) { HANDLE_ERROR( hipEventRecord(stop, 0) ); HANDLE_ERROR( hipEventSynchronize(stop) ); HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) ); time3+=time;} 
        } else if (m->getType() == COPY_TO_CPU) {
            if (timingdump) { printf("4");
            HANDLE_ERROR( hipEventCreate(&start) ); HANDLE_ERROR( hipEventCreate(&stop) ); HANDLE_ERROR( hipEventRecord(start, 0) );}
            for (NameLayerMap::iterator it = _nameLayerMap.begin(); it != _nameLayerMap.end(); ++it) {
                it->second->copyToCPU();
            }
            if (timingdump) { HANDLE_ERROR( hipEventRecord(stop, 0) ); HANDLE_ERROR( hipEventSynchronize(stop) ); HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) ); time4+=time;} 
        } else if (m->getType() == COPY_TO_GPU) {
            if (timingdump) { printf("5");
            HANDLE_ERROR( hipEventCreate(&start) ); HANDLE_ERROR( hipEventCreate(&stop) ); HANDLE_ERROR( hipEventRecord(start, 0) );}
            for (NameLayerMap::iterator it = _nameLayerMap.begin(); it != _nameLayerMap.end(); ++it) {
                it->second->copyToGPU();
            }
            if (timingdump) { HANDLE_ERROR( hipEventRecord(stop, 0) ); HANDLE_ERROR( hipEventSynchronize(stop) ); HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) ); time5+=time;} 
        } else if (m->getType() == RESET) {
            if (timingdump) { printf("6");
            HANDLE_ERROR( hipEventCreate(&start) ); HANDLE_ERROR( hipEventCreate(&stop) ); HANDLE_ERROR( hipEventRecord(start, 0) );}
            for (NameLayerMap::iterator it = _nameLayerMap.begin(); it != _nameLayerMap.end(); ++it) {
                it->second->reset();
            }
            if (timingdump) { HANDLE_ERROR( hipEventRecord(stop, 0) ); HANDLE_ERROR( hipEventSynchronize(stop) ); HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) ); time6+=time;}  
        } else if (m->getType() == RESET_PASS_IDX) {
            if (timingdump) { printf("7");
            HANDLE_ERROR( hipEventCreate(&start) ); HANDLE_ERROR( hipEventCreate(&stop) ); HANDLE_ERROR( hipEventRecord(start, 0) );}
            for (NameLayerMap::iterator it = _nameLayerMap.begin(); it != _nameLayerMap.end(); ++it) {
                it->second->resetPassIdx();
            }
            if (timingdump) { HANDLE_ERROR( hipEventRecord(stop, 0) ); HANDLE_ERROR( hipEventSynchronize(stop) ); HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) ); time7+=time;}
        } else if (m->getType() == UPDATE_WEIGHTS) {
            if (timingdump) { printf("8");
            HANDLE_ERROR( hipEventCreate(&start) ); HANDLE_ERROR( hipEventCreate(&stop) ); HANDLE_ERROR( hipEventRecord(start, 0) );}
            for (NameLayerMap::iterator it = _nameLayerMap.begin(); it != _nameLayerMap.end(); ++it) {
                it->second->updateWeights();
            }
            if (timingdump) { HANDLE_ERROR( hipEventRecord(stop, 0) ); HANDLE_ERROR( hipEventSynchronize(stop) ); HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) ); time8+=time;}
        } else if (m->getType() == CONSTRAIN_WEIGHTS) {
            if (timingdump) { printf("9"); 
            HANDLE_ERROR( hipEventCreate(&start) ); HANDLE_ERROR( hipEventCreate(&stop) ); HANDLE_ERROR( hipEventRecord(start, 0) ); }
            for (NameLayerMap::iterator it = _nameLayerMap.begin(); it != _nameLayerMap.end(); ++it) {
                it->second->constrainWeights();
            }
            if (timingdump) { HANDLE_ERROR( hipEventRecord(stop, 0) ); HANDLE_ERROR( hipEventSynchronize(stop) ); HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) ); time9+=time; }
            // dump all the text here! 
            if (timingdump) { 
                printf("\n------------ timing -------------- \n");
                printf("Time0 to fprop ready:  %3.1f ms \n", time0);
                printf("Time1 to bprop ready:  %3.1f ms \n", time1);
                printf("Time2 to bprop start:  %3.1f ms \n", time2);
                printf("Time3 to sync:  %3.1f ms \n", time3);
                printf("Time4 to copy to cpu:  %3.1f ms \n", time4);
                printf("Time5 to copy to gpu:  %3.1f ms \n", time5);
                printf("Time6 to reset:  %3.1f ms \n", time6);
                printf("Time7 to reset index:  %3.1f ms \n", time7);
                printf("Time8 to update weights:  %3.1f ms \n", time8);
                printf("Time9 to constrain:  %3.1f ms \n", time9);
                time0 = time1 = time2 = time3 = time4 = time5 = time6 = time7 = time8 = time9 = 0;
            }
            // here we get output at every minibatch, great!         
        } else if (m->getType() == EXIT_CONVNET) {
            exit = true;
        }
        delete m;
    }
    return NULL;
}

Cost& ConvNetThread::getCost() {
    // In a single ConvNetThread, all costs are guaranteed to be different
    // (i.e. not replicas of one another)
    return *new Cost(_costs);
}

Layer& ConvNetThread::getLayer(std::string& name) {
    return *_nameLayerMap[name];
}

int ConvNetThread::getDeviceID() {
    return _deviceID;
}

Queue<Message*>& ConvNetThread::getMessageQueue() {
    return _msgQueue;
}

vector<CostLayer*>& ConvNetThread::getCostLayers() {
    return _costs;
}

NameLayerMap& ConvNetThread::getLayerMap() {
    return _nameLayerMap;
}

ConvNet& ConvNetThread::getConvNet() {
    return *_convNet;
}

